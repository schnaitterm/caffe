#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/transpose_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void TransposeForward(
    const int nthreads, const Dtype* bottom_data, const int channels_in,
    const int depth_in, const int height_in, const int width_in,
    const int channels_out, const int height_out,
    const int width_out, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int num = index;
    const int w = num % width_in;
    num = num / width_in;
    const int h = num % height_in;
    num = num / height_in;
    const int d = num % depth_in;
    num = num / depth_in;
    const int c = num % channels_in;
    num = num / channels_in;

    const int out_index = (num * depth_in + d) * channels_out
        + c * height_out
        + h * width_out + w;
    top_data[out_index] = bottom_data[index];
  }
}

template <typename Dtype>
__global__ void TransposeBackward(
    const int nthreads, Dtype* bottom_data, const int channels_in,
    const int depth_in, const int height_in, const int width_in,
    const int channels_out, const int height_out,
    const int width_out, const Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int num = index;
    const int w = num % width_in;
    num = num / width_in;
    const int h = num % height_in;
    num = num / height_in;
    const int d = num % depth_in;
    num = num / depth_in;
    const int c = num % channels_in;
    num = num / channels_in;

    const int out_index = (num * depth_in + d) * channels_out
        + c * height_out
        + h * width_out + w;

    bottom_data[index] = top_data[out_index];
  }
}

template <typename Dtype>
void TransposeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  TransposeForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom[0]->shape(1), bottom[0]->shape(2),
      bottom[0]->shape(3), bottom[0]->shape(4), top[0]->count(1),
      top[0]->count(2), top[0]->count(3), top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void TransposeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  Dtype* bottom_data = bottom[0]->mutable_gpu_diff();
  const Dtype* top_data = top[0]->gpu_diff();
  int count = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  TransposeBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom[0]->shape(1), bottom[0]->shape(2),
      bottom[0]->shape(3), bottom[0]->shape(4), top[0]->count(1),
      top[0]->count(2), top[0]->count(3), top_data);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(TransposeLayer);

}  // namespace caffe
