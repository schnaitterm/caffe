#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/transform2d_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Transform2DForward(
    const int nthreads, const Dtype* bottom_data, const int channels_in,
    const int height_in, const int width_in, const int channels_out,
    const int height_out, const int width_out, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int num = index;
    const int w = num % width_in;
    num = num / width_in;
    const int h = num % height_in;
    num = num / height_in;
    const int c = num % channels_in;
    num = num / channels_in;

    int i = h % 2;
    int out_h = h / 2;
    int j = w % 2;
    int out_w = w / 2;
    int out_c = c + (i * 2 + j) * channels_in;

    const int out_index = num * channels_out + out_c * height_out
        + out_h * width_out + out_w;
    top_data[out_index] = bottom_data[index];
  }
}

template <typename Dtype>
__global__ void Transform2DBackward(
    const int nthreads, Dtype* bottom_data, const int channels_in,
    const int height_in, const int width_in, const int channels_out,
    const int height_out, const int width_out, const Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int num = index;
    const int w = num % width_in;
    num = num / width_in;
    const int h = num % height_in;
    num = num / height_in;
    const int c = num % channels_in;
    num = num / channels_in;

    int i = h % 2;
    int out_h = h / 2;
    int j = w % 2;
    int out_w = w / 2;
    int out_c = c + (i * 2 + j) * channels_in;

    const int out_index = num * channels_out + out_c * height_out
        + out_h * width_out + out_w;
    bottom_data[index] = top_data[out_index];
  }
}

template <typename Dtype>
void Transform2DLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  Transform2DForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom[0]->shape(1), bottom[0]->shape(2),
      bottom[0]->shape(3), top[0]->count(1), top[0]->count(2),
      top[0]->count(3), top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void Transform2DLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  Dtype* bottom_data = bottom[0]->mutable_gpu_diff();
  const Dtype* top_data = top[0]->gpu_diff();
  int count = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  Transform2DBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom[0]->shape(1), bottom[0]->shape(2),
      bottom[0]->shape(3), top[0]->count(1), top[0]->count(2),
      top[0]->count(3), top_data);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(Transform2DLayer);

}  // namespace caffe
