#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/duplicate_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void DuplicateForward(
    const int nthreads, const Dtype* bottom_data, const int channels_in,
    const int height_in, const int width_in, const int channels_out,
    const int height_out, const int width_out, const int duplicates_,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int num = index;
    const int w = num % width_in;
    num = num / width_in;
    const int h = num % height_in;
    num = num / height_in;
    const int c = num % channels_in;
    num = num / channels_in;

    for (int i = 0; i < duplicates_; ++i) {
      const int out_index = (num * duplicates_ + i) * channels_out
           + c * height_out + h * width_out + w;
      top_data[out_index] = bottom_data[index];
    }
  }
}

template <typename Dtype>
__global__ void DuplicateBackward(
    const int nthreads, Dtype* bottom_data, const int channels_in,
    const int height_in, const int width_in, const int channels_out,
    const int height_out, const int width_out, const int duplicates_,
    const Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int num = index;
    const int w = num % width_in;
    num = num / width_in;
    const int h = num % height_in;
    num = num / height_in;
    const int c = num % channels_in;
    num = num / channels_in;

    bottom_data[index] = 0;
    for (int i = 0; i < duplicates_; ++i) {
      const int out_index = (num * duplicates_ + i) * channels_out
           + c * height_out + h * width_out + w;
      bottom_data[index] += top_data[out_index];
    }
  }
}

template <typename Dtype>
void DuplicateLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = bottom[0]->count();

  // NOLINT_NEXT_LINE(whitespace/operators)
  DuplicateForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom[0]->shape(1), bottom[0]->shape(2),
      bottom[0]->shape(3), top[0]->count(1), top[0]->count(2),
      top[0]->count(3), duplicates_, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void DuplicateLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  Dtype* bottom_data = bottom[0]->mutable_gpu_diff();
  const Dtype* top_data = top[0]->gpu_diff();
  int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  DuplicateBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom[0]->shape(1), bottom[0]->shape(2),
      bottom[0]->shape(3), top[0]->count(1), top[0]->count(2),
      top[0]->count(3), duplicates_, top_data);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(DuplicateLayer);

} // namespace caffe
