#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/deconv_trans_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void DeconvTransForward(
    const int nthreads, const Dtype* bottom_data, const int channels_in,
    const int depth_in, const int height_in, const int width_in,
    const int channels_out, const int depth_out, const int height_out,
    const int width_out, const int stride_d, const int stride_h,
    const int stride_w, const int factor, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int num = index;
    const int w = num % width_in;
    num = num / width_in;
    const int h = num % height_in;
    num = num / height_in;
    const int d = num % depth_in;
    num = num / depth_in;
    const int c = num % channels_in;
    num = num / channels_in;

    int pos = c % factor;
    int i = pos % stride_w;
    pos = pos / stride_w;
    int j = pos % stride_h;
    pos = pos / stride_h;
    int k = pos % stride_d;
    int out_c = c / factor;

    const int out_index = num * channels_out + out_c * depth_out
        + (d * stride_d + k) * height_out
        + (h * stride_h + j) * width_out
        + w * stride_w + i;

    top_data[out_index] = bottom_data[index];
  }
}

template <typename Dtype>
__global__ void DeconvTransBackward(
    const int nthreads, Dtype* bottom_data, const int channels_in,
    const int depth_in, const int height_in, const int width_in,
    const int channels_out, const int depth_out, const int height_out,
    const int width_out, const int stride_d, const int stride_h,
    const int stride_w, const int factor, const Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int num = index;
    const int w = num % width_in;
    num = num / width_in;
    const int h = num % height_in;
    num = num / height_in;
    const int d = num % depth_in;
    num = num / depth_in;
    const int c = num % channels_in;
    num = num / channels_in;

    int pos = c % factor;
    int i = pos % stride_w;
    pos = pos / stride_w;
    int j = pos % stride_h;
    pos = pos / stride_h;
    int k = pos % stride_d;
    int out_c = c / factor;

    const int out_index = num * channels_out + out_c * depth_out
        + (d * stride_d + k) * height_out
        + (h * stride_h + j) * width_out
        + w * stride_w + i;

    bottom_data[index] = top_data[out_index];
  }
}

template <typename Dtype>
void DeconvTransLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                          const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  DeconvTransForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom[0]->shape(1), bottom[0]->shape(2),
          bottom[0]->shape(3), bottom[0]->shape(4), top[0]->count(1),
          top[0]->count(2), top[0]->count(3), top[0]->count(4), stride_[0],
          stride_[1], stride_[2], factor_, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void DeconvTransLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                           const vector<bool>& propagate_down,
                                           const vector<Blob<Dtype>*>& bottom) {
  Dtype* bottom_data = bottom[0]->mutable_gpu_diff();
  const Dtype* top_data = top[0]->gpu_diff();
  int count = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  DeconvTransBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom[0]->shape(1), bottom[0]->shape(2),
          bottom[0]->shape(3), bottom[0]->shape(4), top[0]->count(1),
          top[0]->count(2), top[0]->count(3), top[0]->count(4), stride_[0],
          stride_[1], stride_[2], factor_, top_data);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(DeconvTransLayer);

} // namespace caffe