#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/temporal_max_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void temporal_max_forward(const int nthreads, const Dtype* bottom,
                                     const int num, const int count, int* max_idx, Dtype* top) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    max_idx[index] = 0;
    top[index] = bottom[index];
    for (int n = 1; n < num; ++n) {
      int bottom_idx = n * count + index;
      if (bottom[bottom_idx] > top[index]) {
        top[index] = bottom[n * count + index];
        max_idx[index] = n;
      }
    }
  }
}

template <typename Dtype>
void TemporalMaxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                          const vector<Blob<Dtype>*>& top) {
  const int nthreads = bottom[0]->count(1);
  // NOLINT_NEXT_LINE(whitespace/operators)
  temporal_max_forward<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
      nthreads, bottom[0]->gpu_data(), bottom[0]->shape(0), nthreads,
          max_idx_.mutable_gpu_data(), top[0]->mutable_gpu_data());
}

template <typename Dtype>
__global__ void temporal_max_backward(
    const int nthreads, Dtype* bottom, const int num, const int count,
    const int* max_idx, const Dtype* top) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int bottom_index = max_idx[index] * count + index;
    bottom[bottom_index] = top[index];
  }
}

template <typename Dtype>
void TemporalMaxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                           const vector<bool>& propagate_down,
                                           const vector<Blob<Dtype>*>& bottom) {
  const int nthreads = bottom[0]->count(1);
  // NOLINT_NEXT_LINE(whitespace/operators)
  temporal_max_backward<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
      nthreads, bottom[0]->mutable_gpu_diff(), bottom[0]->shape(0), nthreads,
          max_idx_.gpu_data(), top[0]->gpu_diff());
}

INSTANTIATE_LAYER_GPU_FUNCS(TemporalMaxLayer);

} // namespace caffe