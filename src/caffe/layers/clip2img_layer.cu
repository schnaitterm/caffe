#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/clip2img_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void clip2img_forward(const int nthreads, const Dtype* bottom,
    const int num, const int channels, const int depth, const int height,
    const int width, Dtype* top) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int w = index % width;
    const int h = index / width % height;
    const int d = index / width / height % depth;
    const int c = index / width / height / depth % channels;
    const int n = index / width / height / depth / channels;

    int out_index = (n * depth + d) * channels * height * width +
        c * height * width + h * width + w;
    top[out_index] = bottom[index];
  }
}

template <typename Dtype>
void Clip2ImgLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                       const vector<Blob<Dtype>*>& top) {
  const int nthreads = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  clip2img_forward<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
      nthreads, bottom[0]->gpu_data(), bottom_axes_[0],
      bottom_axes_[1], bottom_axes_[2], bottom_axes_[3], bottom_axes_[4],
      top[0]->mutable_gpu_data());
}

template <typename Dtype>
__global__ void clip2img_backward(
    const int nthreads, Dtype* bottom, const int num, const int channels,
    const int depth, const int height, const int width, const Dtype* top) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int w = index % width;
    const int h = index / width % height;
    const int d = index / width / height % depth;
    const int c = index / width / height / depth % channels;
    const int n = index / width / height / depth / channels;

    int top_index = (n * depth + d) * channels * height * width +
        c * height * width + h * width + w;
    bottom[index] = top[top_index];
  }
}

template <typename Dtype>
void Clip2ImgLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                        const vector<bool>& propagate_down,
                                        const vector<Blob<Dtype>*>& bottom) {
  const int nthreads = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  clip2img_backward<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
      nthreads, bottom[0]->mutable_gpu_diff(), bottom_axes_[0],
      bottom_axes_[1], bottom_axes_[2], bottom_axes_[3], bottom_axes_[4],
      top[0]->gpu_diff());
}

INSTANTIATE_LAYER_GPU_FUNCS(Clip2ImgLayer);

} // namespace caffe