#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/normalization_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void NormalizationLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                            const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* squared_data = squared_.mutable_gpu_data();
  Dtype normsqr;
  caffe_gpu_powx(num_item_ * item_count_, bottom_data, Dtype(2), squared_data);
  for (int i = 0; i < num_item_; ++i) {
    caffe_gpu_asum<Dtype>(item_count_, squared_data + i * item_count_, &normsqr);
    caffe_gpu_scale<Dtype>(item_count_, pow(normsqr, -0.5), bottom_data + i * item_count_,
                           top_data + i * item_count_);
  }
}

template <typename Dtype>
void NormalizationLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                             const vector<bool>& propagate_down,
                                             const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  Dtype a;
  for (int i = 0; i < num_item_; ++i) {
    caffe_gpu_dot(item_count_, top_data + i * item_count_,
                  top_diff + i * item_count_, &a);
    caffe_gpu_scale(item_count_, a, top_data + i * item_count_,
                    bottom_diff + i * item_count_);
    caffe_gpu_sub(item_count_, top_diff + i * item_count_,
                  bottom_diff + i * item_count_, bottom_diff + i * item_count_);
    caffe_gpu_dot(item_count_, bottom_data + i * item_count_,
                  bottom_data + i * item_count_, &a);
    caffe_gpu_scale(item_count_, Dtype(pow(a, -0.5)),
                    bottom_diff + i * item_count_,
                    bottom_diff + i * item_count_);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(NormalizationLayer);
} // namespace caffe